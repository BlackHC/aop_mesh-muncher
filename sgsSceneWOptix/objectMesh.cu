#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

rtBuffer<MergedTextureInfo> textureInfos;
rtTextureSampler<float4, 2> objectTexture;

// one per primitive
rtBuffer<int> textureIndices;

struct Vertex
{
	float3 position;
	float3 normal;
	float2 uv[2];
};

rtBuffer<Vertex> vertexBuffer;
rtBuffer<int3> indexBuffer; // position indices

__device__ float4 performTextureLookup() {
	MergedTextureInfo &textureInfo = textureInfos[ textureIndex ];

	const float2 wrappedTexCoords = texCoord - floor( texCoord ); 
	const float2 mergedTexCoords = make_float2( textureInfo.offset ) + wrappedTexCoords * make_float2( textureInfo.size );

	return tex2D( objectTexture, mergedTexCoords.x, mergedTexCoords.y );
}

RT_PROGRAM void closestHit() {
	float3 hitPosition = currentRay.origin + t_hit * currentRay.direction;

	float3 worldShadingNormal   = normalize(shadingNormal);
	float3 worldGeometricNormal = normalize(geometricNormal);
	float3 ffnormal = faceforward(worldShadingNormal, -currentRay.direction, worldGeometricNormal);
	
	// actually -sunDirection but we don't need to care because of the abs
	float diffuseAttenuation = abs( dot( ffnormal, sunDirection ) );

	const float4 textureLookup = performTextureLookup();
	
	const float3 litSurfaceColor = diffuseAttenuation * make_float3( textureLookup ) * getDirectionalLightTransmittance( hitPosition, sunDirection );

	if( textureLookup.w < 1.0 ) {
		// cast another ray
		optix::Ray subRay( hitPosition, currentRay.direction, RT_EYE, sceneEpsilon );
		
		Ray_Eye subRay_eye;
		subRay_eye.color = make_float3( 0.0f );

		rtTrace(rootObject, subRay, subRay_eye);

		currentRay_eye.color = litSurfaceColor * textureLookup.w + subRay_eye.color * (1.0f - textureLookup.w);
	}
	else {
		currentRay_eye.color = litSurfaceColor;
	}
}

RT_PROGRAM void closestHitAdditive() {
	float3 hitPosition = currentRay.origin + t_hit * currentRay.direction;

	const float4 textureLookup = performTextureLookup();
	
	// cast another ray
	optix::Ray subRay( hitPosition, currentRay.direction, RT_EYE, sceneEpsilon );
		
	Ray_Eye subRay_eye;
	subRay_eye.color = make_float3( 0.0f );

	rtTrace(rootObject, subRay, subRay_eye);

	currentRay_eye.color = make_float3( textureLookup ) + subRay_eye.color;
}

RT_PROGRAM void anyHit() {
	const float4 textureLookup = performTextureLookup();
	
	currentRay_shadow.transmittance *= 1.0 - textureLookup.w;
	if( currentRay_shadow.transmittance < 0.01 ) {
		rtTerminateRay();
	}
	else {
		// NOTE: this is important, otherwise it wont take into account other possible hit locations
		rtIgnoreIntersection();
	}
}

// additive objects are fully transparent
RT_PROGRAM void anyHitAdditive() {
	rtIgnoreIntersection();
}

RT_PROGRAM void intersect( int primIdx )
{
	int3 v_idx = indexBuffer[primIdx];

	float3 p0 = vertexBuffer[ v_idx.x ].position;
	float3 p1 = vertexBuffer[ v_idx.y ].position;
	float3 p2 = vertexBuffer[ v_idx.z ].position;

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if( intersect_triangle( currentRay, p0, p1, p2, n, t, beta, gamma ) ) {
		if( rtPotentialIntersection( t ) ) {
			int3 n_idx = indexBuffer[ primIdx ];

			float3 n0 = vertexBuffer[ n_idx.x ].normal;
			float3 n1 = vertexBuffer[ n_idx.y ].normal;
			float3 n2 = vertexBuffer[ n_idx.z ].normal;
			shadingNormal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			geometricNormal = -n;

			int3 t_idx = indexBuffer[ primIdx ];
			
			float2 t0 = vertexBuffer[ t_idx.x ].uv[0];
			float2 t1 = vertexBuffer[ t_idx.y ].uv[0];
			float2 t2 = vertexBuffer[ t_idx.z ].uv[0];
			texCoord = ( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );

			textureIndex = textureIndices[ primIdx ];
				
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void calculateBoundingBox (int primIdx, float result[6]) {
	int3 v_idx = indexBuffer[primIdx];

	float3 v0 = vertexBuffer[ v_idx.x ].position;
	float3 v1 = vertexBuffer[ v_idx.y ].position;
	float3 v2 = vertexBuffer[ v_idx.z ].position;

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf( fminf( v0, v1), v2 );
	aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
}
