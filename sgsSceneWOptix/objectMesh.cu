#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

rtBuffer<MergedTextureInfo> textureInfos;
rtTextureSampler<float4, 2> objectTexture;

// one per primitive
rtBuffer<MaterialInfo> materialInfos;
rtBuffer<int> materialIndices;

struct Vertex
{
	float3 position;
	float3 normal;
	float2 uv[2];
};

rtBuffer<Vertex> vertexBuffer;
rtBuffer<int3> indexBuffer; // position indices

__device__ float4 getTexel() {
	MergedTextureInfo &textureInfo = textureInfos[ materialInfo.textureIndex ];

	const float2 wrappedTexCoords = texCoord - floor( texCoord ); 
	const float2 mergedTexCoords = make_float2( textureInfo.offset ) + wrappedTexCoords * make_float2( textureInfo.size );

	return tex2D( objectTexture, mergedTexCoords.x, mergedTexCoords.y );
}

__device__ float3 subTrace( const float3 &position, const float3 &direction, bool earlyOut ) {
	if( earlyOut ) {
		return make_float3( 0.0 );
	}

	optix::Ray subRay( position, direction, RT_EYE, sceneEpsilon );
		
	Ray_Eye subRay_eye;
	subRay_eye.color = make_float3( 0.0f );

	rtTrace(rootObject, subRay, subRay_eye);
	
	return subRay_eye.color;
}

RT_PROGRAM void closestHit() {
	float3 hitPosition = currentRay.origin + t_hit * currentRay.direction;

	float3 worldShadingNormal   = normalize(shadingNormal);
	float3 worldGeometricNormal = normalize(geometricNormal);
	float3 ffnormal = faceforward(worldShadingNormal, -currentRay.direction, worldGeometricNormal);
	
	// actually -sunDirection but we don't need to care because of the abs
	float diffuseAttenuation = abs( dot( ffnormal, sunDirection ) );

	const float4 diffuseColor = getTexel();
	
	const float3 litSurfaceColor = make_float3( diffuseColor ) * 
		(0.2 + 0.8 * diffuseAttenuation * getDirectionalLightTransmittance( hitPosition, sunDirection ));

	switch( materialInfo.alphaType ) {
	default:
	case MaterialInfo::AT_NONE:
		currentRay_eye.color = litSurfaceColor;
		break;
	case MaterialInfo::AT_ADDITIVE:
		currentRay_eye.color = make_float3( diffuseColor ) + subTrace( hitPosition, currentRay.direction, false );
		break;
	case MaterialInfo::AT_MATERIAL: {
		const float alpha = materialInfo.alpha;
		currentRay_eye.color = litSurfaceColor * alpha + subTrace( hitPosition, currentRay.direction, alpha > 0.99f ) * (1.0f - alpha);
		break;
	}
	case MaterialInfo::AT_TEXTURE:
	case MaterialInfo::AT_ALPHATEST:
		currentRay_eye.color = litSurfaceColor * diffuseColor.w + subTrace( hitPosition, currentRay.direction, diffuseColor.w > 0.99f ) * (1.0f - diffuseColor.w);
		break;
	case MaterialInfo::AT_MULTIPLY:
		currentRay_eye.color = make_float3( diffuseColor ) * subTrace( hitPosition, currentRay.direction, false );
		break;
	case MaterialInfo::AT_MULTIPLY_2:
		currentRay_eye.color = make_float3( diffuseColor ) * subTrace( hitPosition, currentRay.direction, false ) * 2;
		break;
	}
}

RT_PROGRAM void anyHit() {
	switch( materialInfo.alphaType ) {
	default:
	case MaterialInfo::AT_NONE:
		currentRay_shadow.transmittance = 0.0;
		rtTerminateRay();
		return;
	case MaterialInfo::AT_ADDITIVE:
		rtIgnoreIntersection();
		return;
	case MaterialInfo::AT_MATERIAL:
		currentRay_shadow.transmittance *= 1.0 - materialInfo.alpha;
		break;
	case MaterialInfo::AT_ALPHATEST:
	case MaterialInfo::AT_TEXTURE:
		currentRay_shadow.transmittance *= 1.0 - getTexel().w;
		break;
	}

	if( currentRay_shadow.transmittance < 0.01 ) {
		rtTerminateRay();
	}
	else {
		// NOTE: this is important, otherwise it wont take into account other possible hit locations
		rtIgnoreIntersection();
	}
}

RT_PROGRAM void intersect( int primIdx )
{
	int3 v_idx = indexBuffer[primIdx];

	float3 p0 = vertexBuffer[ v_idx.x ].position;
	float3 p1 = vertexBuffer[ v_idx.y ].position;
	float3 p2 = vertexBuffer[ v_idx.z ].position;

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if( intersect_triangle( currentRay, p0, p1, p2, n, t, beta, gamma ) ) {
		if( rtPotentialIntersection( t ) ) {
			int3 n_idx = indexBuffer[ primIdx ];

			float3 n0 = vertexBuffer[ n_idx.x ].normal;
			float3 n1 = vertexBuffer[ n_idx.y ].normal;
			float3 n2 = vertexBuffer[ n_idx.z ].normal;
			shadingNormal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			geometricNormal = -n;

			int3 t_idx = indexBuffer[ primIdx ];
			
			float2 t0 = vertexBuffer[ t_idx.x ].uv[0];
			float2 t1 = vertexBuffer[ t_idx.y ].uv[0];
			float2 t2 = vertexBuffer[ t_idx.z ].uv[0];
			texCoord = ( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );

			materialInfo = materialInfos[ materialIndices[ primIdx ] ];
				
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void calculateBoundingBox (int primIdx, float result[6]) {
	int3 v_idx = indexBuffer[primIdx];

	float3 v0 = vertexBuffer[ v_idx.x ].position;
	float3 v1 = vertexBuffer[ v_idx.y ].position;
	float3 v2 = vertexBuffer[ v_idx.z ].position;

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf( fminf( v0, v1), v2 );
	aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
}
