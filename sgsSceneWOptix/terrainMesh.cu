#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

struct Vertex
{
	float3 position;
	float3 normal;
	float2 uv[1];
};

rtTextureSampler<float4, 2> terrainTexture;
rtBuffer<Vertex> vertexBuffer;
rtBuffer<int3> indexBuffer;

RT_PROGRAM void closestHit()
{
	float3 hitPosition = currentRay.origin + t_hit * currentRay.direction;

	float3 worldShadingNormal   = normalize(shadingNormal);
	float3 worldGeometricNormal = normalize(geometricNormal);
	float3 ffnormal = faceforward(worldShadingNormal, -currentRay.direction, worldGeometricNormal);
	
	// actually -sunDirection but we don't need to care because of the abs
	float diffuseAttenuation = abs( dot( ffnormal, sunDirection ) );

	currentRay_eye.color = make_float3( tex2D( terrainTexture, texCoord.x, texCoord.y ) ) * diffuseAttenuation * getDirectionalLightTransmittance( hitPosition, sunDirection );
}

RT_PROGRAM void anyHit()
{
	currentRay_shadow.transmittance = 0.0;
	rtTerminateRay();
}

RT_PROGRAM void intersect( int primIdx )
{
	int3 v_idx = indexBuffer[primIdx];

	float3 p0 = vertexBuffer[ v_idx.x ].position;
	float3 p1 = vertexBuffer[ v_idx.y ].position;
	float3 p2 = vertexBuffer[ v_idx.z ].position;

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if( intersect_triangle( currentRay, p0, p1, p2, n, t, beta, gamma ) ) {
		if( rtPotentialIntersection( t ) ) {
			int3 n_idx = indexBuffer[ primIdx ];

			float3 n0 = vertexBuffer[ n_idx.x ].normal;
			float3 n1 = vertexBuffer[ n_idx.y ].normal;
			float3 n2 = vertexBuffer[ n_idx.z ].normal;
			shadingNormal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			geometricNormal = -n;

			int3 t_idx = indexBuffer[ primIdx ];
			
			float2 t0 = vertexBuffer[ t_idx.x ].uv[0];
			float2 t1 = vertexBuffer[ t_idx.y ].uv[0];
			float2 t2 = vertexBuffer[ t_idx.z ].uv[0];
			texCoord = ( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void calculateBoundingBox (int primIdx, float result[6])
{
	int3 v_idx = indexBuffer[primIdx];

	float3 v0 = vertexBuffer[ v_idx.x ].position;
	float3 v1 = vertexBuffer[ v_idx.y ].position;
	float3 v2 = vertexBuffer[ v_idx.z ].position;

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf( fminf( v0, v1), v2 );
	aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
}
