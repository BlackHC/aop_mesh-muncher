#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

__device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
						255u);                                                 /* A */
}

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtBuffer<uchar4, 2>	 outputBuffer;

// Camera Params:
rtDeclareVariable(float3, eyePosition, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

RT_PROGRAM void pinholeCamera_rayGeneration()
{
	float2 d = (make_float2(launchIndex) + make_float2(0.5f, 0.5f)) / make_float2(launchDim) * 2.0f - 1.0f;;

	float3 ray_origin = eyePosition;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);
	
	optix::Ray ray( ray_origin, ray_direction, RT_EYE, sceneEpsilon );
	
	Ray_Eye ray_eye;
	ray_eye.color = make_float3( 0.0f );

	rtTrace( rootObject, ray, ray_eye );
		
	outputBuffer[launchIndex] = make_color( ray_eye.color );
}

RT_PROGRAM void exception() {
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		outputBuffer[launchIndex] = make_uchar4(255, 0, 0, 255);
	} else {
		rtPrintExceptionDetails();
	}
}

RT_PROGRAM void miss() {
	currentRay_eye.color = make_float3( 0 );
	currentRay_eye.distance = maxDistance;
}