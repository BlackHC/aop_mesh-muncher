#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

__device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
						255u);                                                 /* A */
}

rtDeclareVariable(uint, selectionRayIndex, rtLaunchIndex, );
rtDeclareVariable(uint, numSelectionRays, rtLaunchDim, );
rtBuffer<SelectionResult> selectionResults;
rtBuffer<float2> selectionRays;

// Camera Params:
rtDeclareVariable(float3, eyePosition, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );



RT_PROGRAM void selectFromPinholeCamera()
{
	float2 selectionRayDirection = selectionRays[ selectionRayIndex ];

	float3 ray_origin = eyePosition;
	float3 ray_direction = normalize(selectionRayDirection.x*U + selectionRayDirection.y*V + W);
	
	optix::Ray ray( ray_origin, ray_direction, RT_SELECTION, sceneEpsilon );
	
	Ray_Selection ray_selection;

	rtTrace( rootObject, ray, ray_selection );

	selectionResults[ selectionRayIndex ] = ray_selection;
}

RT_PROGRAM void selectFromPinholeCamera_exception() {
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		selectionResults[ selectionRayIndex ].modelIndex = selectionResults[ selectionRayIndex ].objectIndex = SelectionResult::SELECTION_INDEX_STACK_OVERFLOW;
	} else {
		rtPrintExceptionDetails();
	}
}

RT_PROGRAM void selection_miss() {
	currentRay_selection.modelIndex = currentRay_selection.objectIndex = SelectionResult::SELECTION_INDEX_MISS;
}