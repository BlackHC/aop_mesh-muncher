#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

__device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
						255u);                                                 /* A */
}

__device__ __inline__ uchar3 make_rgb(const float3& c)
{
	return make_uchar3( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f)); /* R */
}

__device__ __inline__ char3 make_Lab(const float3& floatLab)
{
	return make_char3( static_cast<signed char>(floatLab.x),  /* L */
						static_cast<signed char>(floatLab.y), /* a */
						static_cast<signed char>(floatLab.z) ); /* b */
}

rtDeclareVariable( uint, probeIndex, rtLaunchIndex, );
rtDeclareVariable( uint, numProbes, rtLaunchDim, );
rtDeclareVariable( uint, sampleOffset, , );

#define numHemisphereSamples 39939
rtBuffer<float3> hemisphereSamples;

rtBuffer<Probe> probes;
rtBuffer<ProbeContext> probeContexts;

RT_PROGRAM void sampleProbes() {
	Probe probe = probes[ probeIndex ];

#if 1
	Onb onb( probe.direction );

	//rtPrintf( "%f", dot( onb.m_normal, cross( onb.m_tangent, onb.m_binormal ) ) );

	uint sampleStartIndex = 
			sampleOffset * numProbes * numProbeSamples 
		+ 
			numProbeSamples * probeIndex 
		+ 
			numProbes * 1979
	;

	float avgDistance = 0.0f;
	float3 avgColor = make_float3( 0.0f );
	int numHits = 0;
	for( int rayIndex = 0 ; rayIndex < numProbeSamples ; ++rayIndex ) {
		const float3 sample = hemisphereSamples[ (sampleStartIndex + rayIndex) % numHemisphereSamples ];
		const float3 rayDirection = onb.m_normal * sample.z + onb.m_tangent * sample.x + onb.m_binormal * sample.y;
		
		Ray ray( probe.position, rayDirection, RT_EYE, sceneEpsilon, maxDistance );

		Ray_Eye ray_eye;
		rtTrace( rootObject, ray, ray_eye );

		// TODO: could weight the probes by sample.z
		if( ray_eye.distance < maxDistance ) {
			++numHits;
			avgDistance += ray_eye.distance;
			avgColor += ray_eye.color;
		}
	}
#else
	float avgDistance = 0.0f;
	float3 avgColor = make_float3( 0.0f );
	int numHits = 0;

	{
		Ray ray( probe.position, probe.direction, RT_EYE, sceneEpsilon, maxDistance );

		Ray_Eye ray_eye;
		rtTrace( rootObject, ray, ray_eye );

		if( ray_eye.distance < maxDistance ) {
			++numHits;
			avgDistance += ray_eye.distance;
			avgColor += ray_eye.color;
		}
	}
#endif

	ProbeContext &context = probeContexts[ probeIndex ];
	if( numHits ) {
		avgDistance = avgDistance / numHits;
		avgColor = avgColor / numHits;
	}

	// convert to cielab
	const float3 Lab = CIELAB::fromRGB( avgColor );
	context.Lab = make_Lab( Lab );
	context.distance = avgDistance;
	context.hitCounter = numHits;
}

RT_PROGRAM void sampleProbes_exception() {
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		probeContexts[ probeIndex ].Lab = make_char3( -127, -127, -127 );
		probeContexts[ probeIndex ].hitCounter = 0;
		probeContexts[ probeIndex ].distance = 1.0f;
	} else {
		rtPrintExceptionDetails();
	}
}