#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

__device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
						255u);                                                 /* A */
}

__device__ __inline__ uchar3 make_rgb(const float3& c)
{
	return make_uchar3( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f)); /* R */
}

__device__ __inline__ char3 make_Lab(const float3& floatLab)
{
	return make_char3( static_cast<signed char>(floatLab.x),  /* L */
						static_cast<signed char>(floatLab.y), /* a */
						static_cast<signed char>(floatLab.z) ); /* b */
}

rtDeclareVariable( uint, probeIndex, rtLaunchIndex, );
rtDeclareVariable( uint, numProbes, rtLaunchDim, );
rtDeclareVariable( uint, sampleOffset, , );

#define numHemisphereSamples 39939
rtBuffer<float3> hemisphereSamples;

rtBuffer<TransformedProbe> transformedProbes;
rtBuffer<ProbeSample> probeSamples;

RT_PROGRAM void sampleProbes() {
	TransformedProbe transformedProbe = transformedProbes[ probeIndex ];

#if 0
	Onb onb( transformedProbe.direction );

	//rtPrintf( "%f", dot( onb.m_normal, cross( onb.m_tangent, onb.m_binormal ) ) );

	uint sampleStartIndex =
			sampleOffset * numProbes * numProbeSamples
		+
			numProbeSamples * probeIndex
		+
			numProbes * 1979
	;

	float avgDistance = 0.0f;
	float3 avgColor = make_float3( 0.0f );
	int numHits = 0;
	for( int rayIndex = 0 ; rayIndex < numProbeSamples ; ++rayIndex ) {
		const float3 sample = hemisphereSamples[ (sampleStartIndex + rayIndex) % numHemisphereSamples ];
		const float3 rayDirection = onb.m_normal * sample.z + onb.m_tangent * sample.x + onb.m_binormal * sample.y;

		Ray ray( transformedProbe.position, rayDirection, RT_EYE, sceneEpsilon, maxDistance );

		Ray_Eye ray_eye;
		rtTrace( rootObject, ray, ray_eye );

		// TODO: could weight the probes by sample.z
		if( ray_eye.distance < maxDistance ) {
			++numHits;
			avgDistance += ray_eye.distance;
			avgColor += ray_eye.color;
		}
	}
#else
	float avgDistance = 0.0f;
	float3 avgColor = make_float3( 0.0f );
	int numHits = 0;

	{
		Ray ray( transformedProbe.position, transformedProbe.direction, RT_EYE, sceneEpsilon, maxDistance );

		Ray_Eye ray_eye;
		rtTrace( rootObject, ray, ray_eye );

		if( ray_eye.distance < maxDistance ) {
			++numHits;
			avgDistance += ray_eye.distance;
			avgColor += ray_eye.color;
		}
	}
#endif

	ProbeSample &sample = probeSamples[ probeIndex ];
	if( numHits ) {
		avgDistance = avgDistance / numHits;
		avgColor = avgColor / numHits;
	}

	// convert to cielab
	const float3 colorLab = CIELAB::fromRGB( avgColor );
	sample.colorLab = make_Lab( colorLab );
	sample.distance = avgDistance;
	sample.occlusion = numHits;
}

RT_PROGRAM void sampleProbes_exception() {
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		probeSamples[ probeIndex ].colorLab = make_char3( -127, -127, -127 );
		probeSamples[ probeIndex ].occlusion = 0;
		probeSamples[ probeIndex ].distance = 1.0f;
	} else {
		rtPrintExceptionDetails();
	}
}