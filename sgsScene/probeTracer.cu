#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

__device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
						255u);                                                 /* A */
}

__device__ __inline__ uchar3 make_rgb(const float3& c)
{
	return make_uchar3( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f)); /* R */
}


rtDeclareVariable(uint, probeIndex, rtLaunchIndex, );
rtDeclareVariable(uint, numProbes, rtLaunchDim, );

#define numHemisphereSamples 39939
rtBuffer<float3> hemisphereSamples;

rtBuffer<Probe> probes;
rtBuffer<ProbeContext> probeContexts;

RT_PROGRAM void sampleProbes() {
	Probe probe = probes[ probeIndex ];

	Onb onb( probe.direction );

	//rtPrintf( "%f", dot( onb.m_normal, cross( onb.m_tangent, onb.m_binormal ) ) );

	int sampleStartIndex = numProbeSamples * probeIndex + numProbes * 521;

	float distance = 0.0f;
	float3 color = make_float3( 0.0f );
	int numHits = 0;
	for( int rayIndex = 0 ; rayIndex < numProbeSamples ; ++rayIndex ) {
		const float3 sample = hemisphereSamples[ (sampleStartIndex + rayIndex) % numHemisphereSamples ];
		const float3 rayDirection = onb.m_normal * sample.z + onb.m_tangent * sample.x + onb.m_binormal * sample.y;
		
		Ray ray( probe.position, rayDirection, RT_EYE, sceneEpsilon, maxDistance );

		Ray_Eye ray_eye;
		rtTrace( rootObject, ray, ray_eye );

		// TODO: could weight the probes by sample.z
		if( ray_eye.distance < maxDistance ) {
			++numHits;
			distance += ray_eye.distance;
			color += ray_eye.color;
		}
	}

	ProbeContext &context = probeContexts[ probeIndex ];
	if( numHits ) {
		context.color = make_rgb( color / numHits);
		context.distance = distance / numHits;
	}
	context.hitCounter = numHits;
}

RT_PROGRAM void sampleProbes_exception() {
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		probeContexts[ probeIndex ].color = make_rgb( make_float3( 1.0f, 1.0f, 1.0f ) );
		probeContexts[ probeIndex ].hitCounter = 0;
		probeContexts[ probeIndex ].distance = 1.0f;
	} else {
		rtPrintExceptionDetails();
	}
}