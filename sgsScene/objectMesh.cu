#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

#include "optixProgramInterface.h"

rtBuffer<MergedTextureInfo> textureInfos;
rtTextureSampler<float4, 2> objectTexture;

// one per primitive
rtBuffer<MaterialInfo> materialInfos;
rtBuffer<int> materialIndices;

struct Vertex
{
	float3 position;
	float3 normal;
	float2 uv[1];
};

rtBuffer<Vertex> vertexBuffer;
rtBuffer<int3> indexBuffer; // position indices

__device__ float4 getTexel() {
	MergedTextureInfo &textureInfo = textureInfos[ materialInfo.textureIndex ];

	const float2 wrappedTexCoords = texCoord - floor( texCoord ); 
	const float2 mergedTexCoords = make_float2( textureInfo.offset ) + wrappedTexCoords * make_float2( textureInfo.size );

	return tex2D( objectTexture, mergedTexCoords.x, mergedTexCoords.y );
}

__device__ float3 subTrace( const float3 &position, const float3 &direction, bool earlyOut ) {
	if( earlyOut ) {
		return make_float3( 0.0 );
	}

	optix::Ray subRay( position, direction, RT_EYE, sceneEpsilon );
		
	Ray_Eye subRay_eye;
	subRay_eye.color = make_float3( 0.0f );

	rtTrace(rootObject, subRay, subRay_eye);
	
	return subRay_eye.color;
}

RT_PROGRAM void eye_closestHit() {
	float3 hitPosition = currentRay.origin + t_hit * currentRay.direction;

	currentRay_eye.distance = t_hit;

	float3 worldShadingNormal   = normalize(shadingNormal);
	float3 worldGeometricNormal = normalize(geometricNormal);
	float3 ffnormal = faceforward(worldShadingNormal, -currentRay.direction, worldGeometricNormal);
	
	// actually -sunDirection but we don't need to care because of the abs
	float diffuseAttenuation = abs( dot( ffnormal, sunDirection ) );

	const float4 diffuseColor = getTexel();
	
	const float3 litSurfaceColor = make_float3( diffuseColor ) * 
		(0.2 + 0.8 * diffuseAttenuation * getDirectionalLightTransmittance( hitPosition, sunDirection ));

	switch( materialInfo.alphaType ) {
	default:
	case MaterialInfo::AT_NONE:
		currentRay_eye.color = litSurfaceColor;
		break;
	case MaterialInfo::AT_ADDITIVE:
		currentRay_eye.color = make_float3( diffuseColor ) + subTrace( hitPosition, currentRay.direction, false );
		break;
	case MaterialInfo::AT_MATERIAL: {
		const float alpha = materialInfo.alpha;
		currentRay_eye.color = litSurfaceColor * alpha + subTrace( hitPosition, currentRay.direction, alpha > 0.99f ) * (1.0f - alpha);
		break;
	}
	case MaterialInfo::AT_TEXTURE:
	case MaterialInfo::AT_ALPHATEST:
		currentRay_eye.color = litSurfaceColor * diffuseColor.w + subTrace( hitPosition, currentRay.direction, diffuseColor.w > 0.99f ) * (1.0f - diffuseColor.w);
		break;
	case MaterialInfo::AT_MULTIPLY:
		currentRay_eye.color = make_float3( diffuseColor ) * subTrace( hitPosition, currentRay.direction, false );
		break;
	case MaterialInfo::AT_MULTIPLY_2:
		currentRay_eye.color = make_float3( diffuseColor ) * subTrace( hitPosition, currentRay.direction, false ) * 2;
		break;
	}
}

#if 0
// could be used to avoid subtraces
RT_PROGRAM void eye_anyHit() {
	if( getTexel().w < 0.05 ) {
		rtIgnoreIntersection();
	}
}
#endif

RT_PROGRAM void shadow_anyHit() {
	switch( materialInfo.alphaType ) {
	default:
	case MaterialInfo::AT_NONE:
		currentRay_shadow.transmittance = 0.0;
		rtTerminateRay();
		return;
	case MaterialInfo::AT_ADDITIVE:
		rtIgnoreIntersection();
		return;
	case MaterialInfo::AT_MATERIAL:
		currentRay_shadow.transmittance *= 1.0 - materialInfo.alpha;
		break;
	case MaterialInfo::AT_ALPHATEST:
	case MaterialInfo::AT_TEXTURE:
		currentRay_shadow.transmittance *= 1.0 - getTexel().w;
		break;
	}

	if( currentRay_shadow.transmittance < 0.01 ) {
		rtTerminateRay();
	}
	else {
		// NOTE: this is important, otherwise it wont take into account other possible hit locations
		rtIgnoreIntersection();
	}
}

RT_PROGRAM void selection_anyHit() {
	float alpha = 1.0;
	switch( materialInfo.alphaType ) {
	default:
	case MaterialInfo::AT_NONE:
		return;
	case MaterialInfo::AT_ADDITIVE:
		return;
	case MaterialInfo::AT_MATERIAL:
		alpha = materialInfo.alpha;
		break;
	case MaterialInfo::AT_ALPHATEST:
	case MaterialInfo::AT_TEXTURE:
		alpha = getTexel().w;
		break;
	}

	if( alpha < 0.1 ) {
		rtIgnoreIntersection();
	}
}

RT_PROGRAM void selection_closestHit() {
	currentRay_selection.objectIndex = materialInfo.objectIndex;
	currentRay_selection.modelIndex = materialInfo.modelIndex;

	currentRay_selection.hitPosition = currentRay.origin + t_hit * currentRay.direction;
	currentRay_selection.hitDistance = t_hit;
}

RT_PROGRAM void intersect( int primIdx )
{
	MaterialInfo &minfo = materialInfos[ materialIndices[ primIdx ] ];
	if( minfo.objectIndex == disabledObjectIndex || minfo.modelIndex == disabledModelIndex ) {
		return;
	}

	int3 v_idx = indexBuffer[primIdx];

	float3 p0 = vertexBuffer[ v_idx.x ].position;
	float3 p1 = vertexBuffer[ v_idx.y ].position;
	float3 p2 = vertexBuffer[ v_idx.z ].position;

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if( intersect_triangle( currentRay, p0, p1, p2, n, t, beta, gamma ) ) {
		if( rtPotentialIntersection( t ) ) {
			int3 n_idx = indexBuffer[ primIdx ];

			float3 n0 = vertexBuffer[ n_idx.x ].normal;
			float3 n1 = vertexBuffer[ n_idx.y ].normal;
			float3 n2 = vertexBuffer[ n_idx.z ].normal;
			shadingNormal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			geometricNormal = -n;

			int3 t_idx = indexBuffer[ primIdx ];
			
			float2 t0 = vertexBuffer[ t_idx.x ].uv[0];
			float2 t1 = vertexBuffer[ t_idx.y ].uv[0];
			float2 t2 = vertexBuffer[ t_idx.z ].uv[0];
			texCoord = ( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );

			materialInfo = minfo;

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void calculateBoundingBox (int primIdx, float result[6]) {
	int3 v_idx = indexBuffer[primIdx];

	float3 v0 = vertexBuffer[ v_idx.x ].position;
	float3 v1 = vertexBuffer[ v_idx.y ].position;
	float3 v2 = vertexBuffer[ v_idx.z ].position;

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf( fminf( v0, v1), v2 );
	aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
}
