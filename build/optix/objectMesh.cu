#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

__device__ __inline__ uchar4 make_color(const float3& c)
{
    return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
                        static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
                        static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
                        255u);                                                 /* A */
}

struct Ray_Payload 
{
	float4 result;
};

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtBuffer<uchar4, 2>	 result_buffer;
rtDeclareVariable(rtObject, top_object, , );

// lights
// rtBuffer<float3> LightPosition;
// rtBuffer<float4> LightColor;

// Camera Params:
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );


RT_PROGRAM void ray_gen()
{
	float2 d = (make_float2(launch_index) + make_float2(0.5f, 0.5f)) / make_float2(launch_dim) * 2.0f - 1.0f;;
	result_buffer[launch_index] = make_uchar4 (0, 0, 0, 255);

	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);
    
	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, 0.05f, RT_DEFAULT_MAX);
    
	Ray_Payload payload;
	payload.result = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

	rtTrace(top_object, ray, payload);
	    
	result_buffer[launch_index] = make_color( make_float3( payload.result ) );
}

RT_PROGRAM void exception()
{
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		result_buffer[launch_index] = make_uchar4(255, 0, 0, 255);
	} else {
		rtPrintExceptionDetails();
	}
}

rtDeclareVariable( Ray_Payload, pr_payload, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable( float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, surface_color, attribute surface_color, );

struct PerRayData_occlusion
{
  float occlusion;
};

rtDeclareVariable(PerRayData_occlusion, prd_occlusion, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
RT_PROGRAM void closest_hit()
{
	float3 phit    = ray.origin + t_hit * ray.direction;

	float3 world_shading_normal   = normalize(shading_normal);
	float3 world_geometric_normal = normalize(geometric_normal);
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
		
	pr_payload.result = make_float4( dot( ffnormal, make_float3( 0.0, -1.0, 1.0 ) ) );
}

RT_PROGRAM void any_hit()
{		
	prd_occlusion.occlusion = 1.0f;

	rtTerminateRay();
}

RT_PROGRAM void miss()
{
	pr_payload.result = make_float4 (0 );
}

struct VFormat
{
	float3 position;
	float3 normal;
	float2 uv[1];
};

rtBuffer<VFormat> vertex_buffer;
rtBuffer<int3> index_buffer;				// position indices

RT_PROGRAM void intersect( int primIdx )
{
	int3 v_idx = index_buffer[primIdx];

	float3 p0 = vertex_buffer[ v_idx.x ].position;
	float3 p1 = vertex_buffer[ v_idx.y ].position;
	float3 p2 = vertex_buffer[ v_idx.z ].position;

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {
		if( rtPotentialIntersection( t ) ) {
			int3 n_idx = index_buffer[ primIdx ];

			float3 n0 = vertex_buffer[ n_idx.x ].normal;
			float3 n1 = vertex_buffer[ n_idx.y ].normal;
			float3 n2 = vertex_buffer[ n_idx.z ].normal;
			shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			geometric_normal = -n;

			int3 t_idx = index_buffer[ primIdx ];
			
			float2 t0 = vertex_buffer[ t_idx.x ].uv[0];
			float2 t1 = vertex_buffer[ t_idx.y ].uv[0];
			float2 t2 = vertex_buffer[ t_idx.z ].uv[0];
			texcoord = ( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
				
			/*float3 c0 = vertex_buffer[ t_idx.x ].color;
			float3 c1 = vertex_buffer[ t_idx.y ].color;
			float3 c2 = vertex_buffer[ t_idx.z ].color;*/
			surface_color = make_float3( 1.0 ); //c1*beta + c2*gamma + c0*(1.0f - beta-gamma);

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void bounding_box (int primIdx, float result[6])
{
	int3 v_idx = index_buffer[primIdx];

	float3 v0 = vertex_buffer[ v_idx.x ].position;
	float3 v1 = vertex_buffer[ v_idx.y ].position;
	float3 v2 = vertex_buffer[ v_idx.z ].position;

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf( fminf( v0, v1), v2 );
	aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
}
