#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

struct Ray_Payload 
{
	float4 result;
};

rtDeclareVariable( Ray_Payload, pr_payload, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable( float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, surface_color, attribute surface_color, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float,      t_hit, rtIntersectionDistance, );

RT_PROGRAM void closest_hit()
{
	float3 phit    = ray.origin + t_hit * ray.direction;

	float3 world_shading_normal   = normalize(shading_normal);
	float3 world_geometric_normal = normalize(geometric_normal);
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
		
	pr_payload.result = make_float4( make_float3( abs( dot( ffnormal, make_float3( 0.0, 1.0, 1.0 ) ) ) ), 1.0 );
}

RT_PROGRAM void any_hit()
{		
	rtTerminateRay();
}

struct VFormat
{
	float3 position;
	float3 normal;
	float2 uv[2];
};

rtBuffer<VFormat> vertex_buffer;
rtBuffer<int3> index_buffer; // position indices

RT_PROGRAM void intersect( int primIdx )
{
	int3 v_idx = index_buffer[primIdx];

	float3 p0 = vertex_buffer[ v_idx.x ].position;
	float3 p1 = vertex_buffer[ v_idx.y ].position;
	float3 p2 = vertex_buffer[ v_idx.z ].position;

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {
		if( rtPotentialIntersection( t ) ) {
			int3 n_idx = index_buffer[ primIdx ];

			float3 n0 = vertex_buffer[ n_idx.x ].normal;
			float3 n1 = vertex_buffer[ n_idx.y ].normal;
			float3 n2 = vertex_buffer[ n_idx.z ].normal;
			shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
			geometric_normal = -n;

			int3 t_idx = index_buffer[ primIdx ];
			
			float2 t0 = vertex_buffer[ t_idx.x ].uv[0];
			float2 t1 = vertex_buffer[ t_idx.y ].uv[0];
			float2 t2 = vertex_buffer[ t_idx.z ].uv[0];
			texcoord = ( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
				
			/*float3 c0 = vertex_buffer[ t_idx.x ].color;
			float3 c1 = vertex_buffer[ t_idx.y ].color;
			float3 c2 = vertex_buffer[ t_idx.z ].color;*/
			surface_color = make_float3( 1.0 ); //c1*beta + c2*gamma + c0*(1.0f - beta-gamma);

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void bounding_box (int primIdx, float result[6])
{
	int3 v_idx = index_buffer[primIdx];

	float3 v0 = vertex_buffer[ v_idx.x ].position;
	float3 v1 = vertex_buffer[ v_idx.y ].position;
	float3 v2 = vertex_buffer[ v_idx.z ].position;

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf( fminf( v0, v1), v2 );
	aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
}
