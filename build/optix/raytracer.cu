#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

__device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
						static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
						static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
						255u);                                                 /* A */
}

struct Ray_Payload 
{
	float4 result;
};

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtBuffer<uchar4, 2>	 result_buffer;
rtDeclareVariable(rtObject, top_object, , );

// lights
// rtBuffer<float3> LightPosition;
// rtBuffer<float4> LightColor;

// Camera Params:
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable( Ray_Payload, pr_payload, rtPayload, );

RT_PROGRAM void ray_gen()
{
	float2 d = (make_float2(launch_index) + make_float2(0.5f, 0.5f)) / make_float2(launch_dim) * 2.0f - 1.0f;;
	result_buffer[launch_index] = make_uchar4 (0, 0, 0, 255);

	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);
	
	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, 0.05f, RT_DEFAULT_MAX);
	
	Ray_Payload payload;
	payload.result = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

	rtTrace(top_object, ray, payload);
		
	result_buffer[launch_index] = make_color( make_float3( payload.result ) );
}

RT_PROGRAM void exception()
{
	unsigned int const error_code = rtGetExceptionCode();
	if(RT_EXCEPTION_STACK_OVERFLOW == error_code) {
		result_buffer[launch_index] = make_uchar4(255, 0, 0, 255);
	} else {
		rtPrintExceptionDetails();
	}
}

RT_PROGRAM void miss()
{
	pr_payload.result = make_float4 (0 );
}